#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "util.cuh"
#include "util/cuda_info.hpp"

#include <cutlass/half.h>

#include <fstream>
#include <iomanip>

#define CHECK_CORRECTNESS true

//// =============== Test Config=============== ////
static const int kWarpPerRow = 2;
static const int kWarpPerCol = 4;
using WholeShape = GemmShape<4096, 4096, 4096>;
using CtaTileShape = GemmShape<128, 128, 64>;
using WarpLayout = tl::RowMajor<kWarpPerRow, kWarpPerCol>;
static constexpr int kRK = 16;

void run_test(std::ofstream& fout) {
    //// =============== Declaration =============== ////
    static constexpr int kM = dim_size<0, WholeShape>;
    static constexpr int kN = dim_size<1, WholeShape>;
    static constexpr int kK = dim_size<2, WholeShape>;

    static constexpr int kTM = dim_size<0, CtaTileShape>;
    static constexpr int kTN = dim_size<1, CtaTileShape>;
    static constexpr int kTK = dim_size<2, CtaTileShape>;

    using InType = cutlass::half_t;
    using AccType = float;

    using Config = KeGemmTraits<InType, AccType, WholeShape, CtaTileShape, kRK,
                                WarpLayout>;
    auto tilefusion_gemm =
        &gemm<InType, kM, kN, kK, kTM, kTN, kTK, typename Config::SIteratorA,
              typename Config::SharedA, typename Config::RegA,
              typename Config::G2SLoaderA, typename Config::S2RLoaderA,
              typename Config::SIteratorB, typename Config::SharedB,
              typename Config::RegB, typename Config::G2SLoaderB,
              typename Config::S2RLoaderB, typename Config::GlobalC,
              typename Config::SharedC, typename Config::Acc,
              typename Config::AccHalf, typename Config::CastAcc,
              typename Config::R2SStorerC, typename Config::S2GStorerC>;

    using KeTraits = benchmarks::cutlass_wrapper::GemmTraits<
        InType, kWarpPerRow, kWarpPerCol, kM, kN, kK, kTM, kTN, kTK>;
    auto cutlass_gemm =
        &benchmarks::cutlass_wrapper::gemm_kernel<InType, kM, kN, kK, kTM, kTN,
                                                  kTK, KeTraits>;

    static constexpr int inputs = kTK * (kTN + kTM) * sizeof(InType);
    static constexpr int acc = kTM * kTN * sizeof(InType);
    static constexpr int smem_size = inputs > acc ? inputs : acc;

    const int kMaxSmemPerBlock = 48 * 1024;
    if (smem_size > kMaxSmemPerBlock) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(tilefusion_gemm),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             smem_size);
        hipFuncSetAttribute(reinterpret_cast<const void*>(cutlass_gemm),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             smem_size);
    }

    int block_x = benchmarks::CeilDiv<kM, kTM>;
    int block_y = benchmarks::CeilDiv<kN, kTN>;
    dim3 dim_grid(block_x, block_y, 1);
    dim3 dim_block(Config::kThreads, 1, 1);

    std::cout << "Running test:" << std::endl
              << "[M, N, K] = " << kM << ", " << kN << ", " << kK
              << ", [TM, TN, TK] = " << kTM << ", " << kTN << ", " << kTK
              << ", RK = " << kRK << ", WarpLayout = [" << kWarpPerRow << ", "
              << kWarpPerCol << "]" << std::endl
              << "blocks = [" << block_x << ", " << block_y << "]" << std::endl
              << std::endl;

    //// =============== Prepare data =============== ////
    // input matrix A
    thrust::host_vector<InType> h_a(kM * kK);
    for (int i = 0; i < h_a.size(); ++i)
        h_a[i] = static_cast<InType>(rand_float());
    thrust::device_vector<InType> d_a = h_a;
    const InType* dA = thrust::raw_pointer_cast(d_a.data());
    const __half* dA2 = reinterpret_cast<const __half*>(dA);

    // input matrix B
    thrust::host_vector<InType> h_b(kK * kN);
    for (int i = 0; i < h_b.size(); ++i)
        h_b[i] = static_cast<InType>(rand_float());
    thrust::device_vector<InType> d_b = h_b;
    const InType* dB = thrust::raw_pointer_cast(d_b.data());
    const __half* dB2 = reinterpret_cast<const __half*>(dB);

    // output matrix C for cutlass GEMM kernel
    thrust::device_vector<InType> d_c(kM * kN);
    InType* dC = thrust::raw_pointer_cast(d_c.data());

    // output matrix C for tilefusion gemm kernel
    thrust::device_vector<InType> d_c2(kM * kN);
    InType* dC2 = thrust::raw_pointer_cast(d_c2.data());

    // output matrix C for cublas gemm
    thrust::device_vector<__half> d_c3(kM * kN);
    __half* dC3 = thrust::raw_pointer_cast(d_c3.data());

    thrust::host_vector<InType> h_c;
    thrust::host_vector<InType> h_c2;
    thrust::host_vector<__half> h_c3;

    //// =============== check correctness =============== ////
#ifdef CHECK_CORRECTNESS
    thrust::fill(d_c.begin(), d_c.end(), static_cast<InType>(0.));
    thrust::fill(d_c2.begin(), d_c2.end(), static_cast<InType>(0.));
    thrust::fill(d_c3.begin(), d_c3.end(), static_cast<__half>(0.));

    cutlass_gemm<<<dim_grid, dim_block, smem_size>>>(dA, dB, dC);
    hipDeviceSynchronize();
    h_c = d_c;

    tilefusion_gemm<<<dim_grid, dim_block, smem_size>>>(dA, dB, dC2);
    hipDeviceSynchronize();
    h_c2 = d_c2;

    // cublas
    cublas_hgemm(kM, kN, kK, dA2, dB2, dC3, false /*timeit*/);
    h_c3 = d_c3;

    bool passed1 = check_results(
        thrust::raw_pointer_cast(h_c.data()) /*cutlass*/,
        thrust::raw_pointer_cast(h_c2.data()) /*tiled cuda*/, kM * kN);

    bool passed2 = check_results(
        thrust::raw_pointer_cast(h_c3.data()) /*cutlass*/,
        thrust::raw_pointer_cast(h_c2.data()) /*tiled cuda*/, kM * kN);

    if (!(passed1 && passed2)) {
        std::cerr << "Test failed" << std::endl;
        return;
    }
    std::cout << "Test passed" << std::endl;
#endif

    //// =============== Timing =============== ////
    thrust::fill(d_c.begin(), d_c.end(), static_cast<InType>(0.));
    thrust::fill(d_c2.begin(), d_c2.end(), static_cast<InType>(0.));
    thrust::fill(d_c3.begin(), d_c3.end(), static_cast<__half>(0.));

    float cublas_time = cublas_hgemm(kM, kN, kK, dA2, dB2, dC3, true);
    h_c3 = d_c3;

    const int warm_up = 10;
    const int iters = 50;
    for (int i = 0; i < warm_up; ++i) {
        cutlass_gemm<<<dim_grid, dim_block, smem_size>>>(dA, dB, dC);
        tilefusion_gemm<<<dim_grid, dim_block, smem_size>>>(dA, dB, dC2);
    }
    hipDeviceSynchronize();

    CudaTimer timer;
    timer.start();
    for (int i = 0; i < iters; ++i) {
        cutlass_gemm<<<dim_grid, dim_block, smem_size>>>(dA, dB, dC);
    }
    hipDeviceSynchronize();
    float cutlass_time = timer.stop() / iters;

    timer.start();
    for (int i = 0; i < iters; ++i) {
        tilefusion_gemm<<<dim_grid, dim_block, smem_size>>>(dA, dB, dC2);
    }
    hipDeviceSynchronize();
    float tilefusion_time = timer.stop() / iters;

    float base = cublas_time;

    fout << "[" << kM << ", " << kN << ", " << kK << "]\t[" << kTM << ", "
         << kTN << ", " << kTK << "]\t" << kRK << "\t[" << kWarpPerRow << ", "
         << kWarpPerCol << "]\t" << cublas_time << "\t" << base << "("
         << std::setprecision(2) << cutlass_time / base << ")"
         << "\t" << std::setprecision(6) << tilefusion_time << " ("
         << std::setprecision(2) << tilefusion_time / base << ")" << std::endl;
}

int main() {
    std::ofstream fout;
    fout.setf(std::ios::fixed);
    fout.precision(6);

    auto dev_name = tilefusion::get_device_name();
    std::stringstream file_name;
    file_name << "figures/bench_" << dev_name << "_gemm.tsv";
    fout.open(file_name.str(), std::ios::out);

    fout << "[M, N, K]\t[kTM, kTN, kTK]\tkRK\tWarp Layout\t"
            "cuBLAS(ms)\tcutlass(ms)\ttilefusion(ms)"
         << std::endl;

    run_test(fout);
    return 0;
}
