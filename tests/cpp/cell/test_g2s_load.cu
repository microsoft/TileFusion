#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cell/mod.hpp"
#include "common/test_utils.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace tilefusion::testing {
using namespace cell;
using namespace copy::warp;
namespace tl = tile_layout;

namespace {
template <typename Element, typename SrcTile, typename DstTile, typename Loader,
          typename Storer>
__global__ void copy_g2s(const Element* src_ptr, Element* dst_ptr,
                         Loader& loader, Storer& storer) {
    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* buf = reinterpret_cast<Element*>(buf_);

    SrcTile src(src_ptr);  // global memory tile
    DstTile inter(buf);    // shared memory tile
    SrcTile dst(dst_ptr);  // global memory tile

    loader(src, inter);
    copy::__copy_async();
    __syncthreads();

    storer(inter, dst);
    __syncthreads();

#if defined(DEBUG)
    if (thread(0)) {
        printf("\nshared\n");
        inter.dump_value();

        printf("\nglobal\n");
        dst.dump_value();
        printf("\n");
    }
#endif
}

template <typename Element, typename WarpLayout, const int kRows,
          const int kCols, const bool kSwizzled = false>
void run_test_row_major() {
    static const int kThreads = tl::get_numel<WarpLayout> * 32;

    int numel = kRows * kCols;
    thrust::host_vector<Element> h_A(numel);
    for (int i = 0; i < h_A.size(); ++i)
        h_A[i] = static_cast<Element>(i % 2048);

    thrust::device_vector<Element> d_B(numel);
    thrust::fill(d_B.begin(), d_B.end(), static_cast<Element>(0.));
    thrust::device_vector<Element> d_A = h_A;

    using SrcTile = GlobalTile<Element, tl::RowMajor<kRows, kCols>>;
    using DstTile = SharedTile<Element, tl::RowMajor<kRows, kCols>, kSwizzled>;

    using Loader = copy::GlobalToSharedLoader<DstTile, WarpLayout>;
    Loader loader;

    using Storer = copy::SharedToGlobalStorer<DstTile, WarpLayout>;
    Storer storer;

    auto copy_kernel = copy_g2s<Element, SrcTile, DstTile, Loader, Storer>;

    int shm_size = kRows * kCols * sizeof(Element);
    if (shm_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            copy_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    dim3 dim_grid(1, 1);
    dim3 dim_block(kThreads);
    copy_kernel<<<dim_grid, dim_block, shm_size>>>(
        thrust::raw_pointer_cast(d_A.data()),
        thrust::raw_pointer_cast(d_B.data()), loader, storer);
    hipDeviceSynchronize();

    thrust::host_vector<Element> h_B(numel);
    h_B = d_B;

    assert_equal(
        reinterpret_cast<Element*>(thrust::raw_pointer_cast(h_A.data())),
        reinterpret_cast<Element*>(thrust::raw_pointer_cast(h_B.data())), numel,
        1e-5);
}

template <typename Element, typename WarpLayout, const int kRows,
          const int kCols, const bool kSwizzled = false>
void run_test_col_major() {
    static const int kThreads = tl::get_numel<WarpLayout> * 32;

    int numel = kRows * kCols;
    thrust::host_vector<Element> h_A(numel);
    for (int i = 0; i < h_A.size(); ++i)
        h_A[i] = static_cast<Element>(i % 2048);

    thrust::device_vector<Element> d_B(numel);
    thrust::fill(d_B.begin(), d_B.end(), static_cast<Element>(0.));
    thrust::device_vector<Element> d_A = h_A;

    using SrcTile = GlobalTile<Element, tl::ColMajor<kRows, kCols>>;
    using DstTile = SharedTile<Element, tl::ColMajor<kRows, kCols>, kSwizzled>;

    using Loader = copy::GlobalToSharedLoader<DstTile, WarpLayout>;
    Loader loader;

    using Storer = copy::SharedToGlobalStorer<DstTile, WarpLayout>;
    Storer storer;

    dim3 dim_grid(1, 1);
    dim3 dim_block(kThreads);

    auto copy_kernel = copy_g2s<Element, SrcTile, DstTile, Loader, Storer>;

    int shm_size = kRows * kCols * sizeof(Element);
    if (shm_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            copy_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    copy_kernel<<<dim_grid, dim_block, shm_size>>>(
        thrust::raw_pointer_cast(d_A.data()),
        thrust::raw_pointer_cast(d_B.data()), loader, storer);
    hipDeviceSynchronize();

    thrust::host_vector<Element> h_B(numel);
    h_B = d_B;

    assert_equal(
        reinterpret_cast<Element*>(thrust::raw_pointer_cast(h_A.data())),
        reinterpret_cast<Element*>(thrust::raw_pointer_cast(h_B.data())), numel,
        1e-5);
}
}  // namespace

TEST(GlobalToSharedLoad, test_row_major_load) {
    {  // test non-swizzled __half.
        static constexpr bool kSwizzled = false;

        run_test_row_major<__half, tl::RowMajor<1, 1>, 16, 64, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<1, 1>, 16, 256, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 1>, 64, 64, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<1, 4>, 16, 256, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<4, 1>, 64, 64, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 2>, 32, 128, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 4>, 32, 256, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 4>, 64, 512, kSwizzled>();
    }

    {  // test swizzled __half.
        static constexpr bool kSwizzled = true;

        run_test_row_major<__half, tl::RowMajor<1, 1>, 16, 64, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<1, 1>, 16, 256, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 1>, 64, 64, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<1, 4>, 16, 256, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<4, 1>, 64, 64, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 2>, 32, 128, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 4>, 32, 256, kSwizzled>();
        run_test_row_major<__half, tl::RowMajor<2, 4>, 64, 512, kSwizzled>();
    }

    {  // test non-swizzled float.
        static constexpr bool kSwizzled = false;

        run_test_row_major<float, tl::RowMajor<1, 1>, 8, 32, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<1, 1>, 16, 64, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<1, 4>, 16, 128, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<4, 1>, 64, 32, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<2, 2>, 32, 64, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<2, 4>, 32, 128, kSwizzled>();
    }

    {  // test swizzled float.
        static constexpr bool kSwizzled = true;

        run_test_row_major<float, tl::RowMajor<1, 1>, 8, 32, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<1, 1>, 16, 64, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<1, 4>, 16, 128, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<4, 1>, 64, 32, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<2, 2>, 32, 64, kSwizzled>();
        run_test_row_major<float, tl::RowMajor<2, 4>, 32, 128, kSwizzled>();
    }
}

TEST(GlobalToSharedLoad, test_col_major_load) {
    {
        static constexpr bool kSwizzled = false;

        run_test_col_major<__half, tl::RowMajor<1, 1>, 64, 16, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<1, 1>, 128, 16, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<1, 4>, 64, 128, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<4, 1>, 256, 16, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<2, 2>, 128, 32, kSwizzled>();
    }

    {
        static constexpr bool kSwizzled = true;
        run_test_col_major<__half, tl::RowMajor<1, 1>, 64, 16, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<1, 1>, 128, 16, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<1, 4>, 64, 128, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<4, 1>, 256, 32, kSwizzled>();
        run_test_col_major<__half, tl::RowMajor<2, 2>, 128, 32, kSwizzled>();
    }

    {
        static constexpr bool kSwizzled = false;

        run_test_col_major<float, tl::RowMajor<1, 1>, 32, 16, kSwizzled>();
        run_test_col_major<float, tl::RowMajor<1, 1>, 64, 16, kSwizzled>();
        run_test_col_major<float, tl::RowMajor<1, 4>, 64, 64, kSwizzled>();
        run_test_col_major<float, tl::RowMajor<4, 1>, 128, 32, kSwizzled>();
        run_test_col_major<float, tl::RowMajor<2, 2>, 64, 64, kSwizzled>();
    }

    {
        static constexpr bool kSwizzled = true;
        run_test_col_major<float, tl::RowMajor<1, 1>, 128, 128, kSwizzled>();
    }
}
}  // namespace tilefusion::testing
