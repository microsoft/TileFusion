// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "common/test_utils.hpp"
#include "types/mod.hpp"

#include <thrust/host_vector.h>

namespace tilefusion::testing {

using namespace cell;
namespace tl = tile_layout;

namespace {
template <typename Layout, typename ChunkShape, const tl::Layout kType>
struct GTileIteratorTester;

template <typename Layout_, typename ChunkShape>
struct GTileIteratorTester<Layout_, ChunkShape, tl::Layout::kRowMajor> {
    using Element = float;
    using Layout = Layout_;

    static constexpr int kRows = Layout::kRows;
    static constexpr int kCols = Layout::kCols;

    static constexpr int kStride0 = dim_size<0, ChunkShape>;
    static constexpr int kStride1 = dim_size<1, ChunkShape>;

    const int kTileRowStride = kStride0 * Layout::kRowStride;
    const int kTileColStride = kStride1;

    static_assert(kRows % kStride0 == 0, "kRows must be divisible by kStride0");
    static_assert(kCols % kStride1 == 0, "kCols must be divisible by kStride1");

    using Tile = GlobalTile<Element, tl::RowMajor<kRows, kCols>>;
    using Iterator = GTileIterator<Tile, ChunkShape>;

    void operator()() {
        int numel = kRows * kCols;
        thrust::host_vector<Element> data(numel);

        Layout layout;
        Element* ptr = data.data();
        int count = 0;
        for (int i = 0; i < kRows; ++i)
            for (int j = 0; j < kCols; ++j) ptr[count++] = layout(i, j);

#if defined(DEBUG_PRINT)
        Tile gtile(ptr);
        gtile.dump_value();
#endif

        EXPECT_EQ(Iterator::sc0, kRows / kStride0);
        EXPECT_EQ(Iterator::sc1, kCols / kStride1);

        Iterator iter(data.data());

        for (int i = 0; i < Iterator::sc0; ++i) {
            for (int j = 0; j < Iterator::sc1; ++j) {
                int start_n = i * kTileRowStride + j * kTileColStride;
                auto tile = iter(i, j);
                for (int m = 0; m < kStride0; ++m) {
                    for (int n = 0; n < kStride1; ++n) {
                        int v1 = int(tile(m, n));
                        int v2 = start_n + m * Layout::kRowStride + n;
                        EXPECT_EQ(v1, v2);
                    }
                }

#if defined(DEBUG_PRINT)
                printf("\nIteration-[%d, %d]:\n", i, j);
                iter(i, j).dump_value();
                printf("\n");
#endif
            }
        }
    }
};

template <typename Layout_, typename ChunkShape>
struct GTileIteratorTester<Layout_, ChunkShape, tl::Layout::kColMajor> {
    using Element = float;
    using Layout = Layout_;

    static constexpr int kRows = Layout::kRows;
    static constexpr int kCols = Layout::kCols;

    static constexpr int kStride0 = dim_size<0, ChunkShape>;
    static constexpr int kStride1 = dim_size<1, ChunkShape>;

    const int kTileRowStride = kStride0;
    const int kTileColStride = kStride1 * Layout::kColStride;

    static_assert(kRows % kStride0 == 0, "kRows must be divisible by kStride0");
    static_assert(kCols % kStride1 == 0, "kCols must be divisible by kStride1");

    using Tile = GlobalTile<Element, tl::RowMajor<kRows, kCols>>;
    using Iterator = GTileIterator<Tile, ChunkShape>;

    void operator()() {
        int numel = kRows * kCols;
        thrust::host_vector<Element> data(numel);

        Layout layout;
        Element* ptr = data.data();
        int count = 0;
        for (int i = 0; i < kRows; ++i)
            for (int j = 0; j < kCols; ++j) ptr[count++] = layout(i, j);

#if defined(DEBUG_PRINT)
        Tile gtile(ptr);
        gtile.dump_value();
#endif

        EXPECT_EQ(Iterator::sc0, kRows / kStride0);
        EXPECT_EQ(Iterator::sc1, kCols / kStride1);

        Iterator iter(data.data());

        for (int i = 0; i < Iterator::sc0; ++i) {
            for (int j = 0; j < Iterator::sc1; ++j) {
                int start_n = i * kTileRowStride + j * kTileColStride;

                auto tile = iter(i, j);
                for (int m = 0; m < kStride0; ++m) {
                    for (int n = 0; n < kStride1; ++n) {
                        int v1 = int(tile(m, n));
                        int v2 = start_n + m + n * Layout::kColStride;

                        EXPECT_EQ(v1, v2);
                    }
                }

#if defined(DEBUG_PRINT)
                printf("\nIteration-[%d, %d]:\n", i, j);
                iter(i, j).dump_value();
                printf("\n");
#endif
            }
        }
    }
};
}  // namespace

TEST(TestGTileIterator, test_row_major) {
    using Tester = GTileIteratorTester<tl::RowMajor<4, 9>, TileShape<2, 3>,
                                       tl::Layout::kRowMajor>;
    Tester tester;
    tester();
}

TEST(TestGTileIterator, col_major) {
    using Tester = GTileIteratorTester<tl::ColMajor<4, 9>, TileShape<2, 3>,
                                       tl::Layout::kColMajor>;
    Tester tester;
    tester();
}
}  // namespace tilefusion::testing
