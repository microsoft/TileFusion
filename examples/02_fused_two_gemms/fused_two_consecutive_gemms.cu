#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "kernels/fused_two_gemms.hpp"
#include "util.hpp"

using namespace tilefusion::kernels;
namespace tl = tilefusion::tile_layout;

template <typename WholeShape, typename CtaTileShape, typename WarpLayout,
          const int kBatch, const int kSharedAccess>
void run(float epsilon = 1e-3) {
    using InType = __half;
    using AccType = float;

    static constexpr int kM = dim_size<0, WholeShape>;
    static constexpr int kN = dim_size<1, WholeShape>;
    static constexpr int kK = dim_size<2, WholeShape>;
    static constexpr int kP = dim_size<3, WholeShape>;

    static constexpr int kTM = dim_size<0, CtaTileShape>;
    static constexpr int kTN = dim_size<1, CtaTileShape>;
    static constexpr int kTK = dim_size<2, CtaTileShape>;
    static constexpr int kTP = dim_size<3, CtaTileShape>;

    static_assert(kK == kTK, "The current implementation requires kTK == K.");
    static_assert(kP == kTP, "The current implementation requires kTP == P.");

    thrust::host_vector<InType> h_a(kM * kK * kBatch);

    for (int i = 0; i < h_a.size(); ++i) {
        h_a[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_b(kK * kN * kBatch);
    for (int i = 0; i < h_b.size(); ++i) {
        h_b[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_c(kN * kP * kBatch);
    for (int i = 0; i < h_c.size(); ++i) {
        h_c[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_d(kM * kP * kBatch);
    thrust::fill(h_d.begin(), h_d.end(), 0.);

    thrust::device_vector<InType> d_a = h_a;
    thrust::device_vector<InType> d_b = h_b;
    thrust::device_vector<InType> d_c = h_c;
    thrust::device_vector<InType> d_d = h_d;

    const InType* A = thrust::raw_pointer_cast(d_a.data());
    const InType* B = thrust::raw_pointer_cast(d_b.data());
    const InType* C = thrust::raw_pointer_cast(d_c.data());
    InType* D = thrust::raw_pointer_cast(d_d.data());

    using Config = FusedTwoGemmsTraits<InType, AccType, WholeShape,
                                       CtaTileShape, WarpLayout, kSharedAccess>;

    using RegA = typename Config::RegA;
    using RegB = typename Config::RegB;
    using RegC = typename Config::RegC;
    using RegD = typename Config::RegD;
    using RegDHalf = typename Config::RegDHalf;
    using RegAcc = typename Config::RegAcc;
    using RegAccCast = typename Config::RegAccCast;

    using GIteratorA = typename Config::GIteratorA;
    using SharedA = typename Config::SharedA;
    using SharedALoader = typename Config::SharedALoader;
    using RegALoader = typename Config::RegALoader;

    using GIteratorB = typename Config::GIteratorB;
    using SharedB = typename Config::SharedB;
    using SharedBLoader = typename Config::SharedBLoader;
    using RegBLoader = typename Config::RegBLoader;

    using GIteratorC = typename Config::GIteratorC;
    using SharedC = typename Config::SharedC;
    using SharedCLoader = typename Config::SharedCLoader;
    using RegCLoader = typename Config::RegCLoader;

    using SharedD = typename Config::SharedD;
    using StoreRegD = typename Config::StoreRegD;
    using StoreSharedD = typename Config::StoreSharedD;

    using ConvertAcc = typename Config::ConvertHalf;
    using ConvertD = typename Config::ConvertD;

    int block_x = CeilDiv<kM, kTM>;
    int block_y = CeilDiv<kP, kTP>;
    int block_z = kBatch;

    dim3 grid(block_x, block_y, block_z);
    dim3 block(Config::kThreads, 1, 1);

    int shm_input = (kTM * kTK + kTK * kTN + kTN * kTP);
    int shm_output = kTM * kTP;
    int shm_size = shm_input < shm_output ? shm_output * sizeof(InType)
                                          : shm_input * sizeof(InType);

    auto kernel =
        &ke_fused_two_gemms<InType, AccType,            //
                            GIteratorA, SharedA, RegA,  //
                            SharedALoader, RegALoader,  //
                            GIteratorB, SharedB, RegB,  //
                            SharedBLoader, RegBLoader,  //
                            GIteratorC, SharedC, RegC,  //
                            SharedCLoader, RegCLoader,  //
                            RegAcc, RegAccCast, typename Config::GlobalD,
                            SharedD, RegD, RegDHalf, StoreRegD, StoreSharedD,
                            ConvertAcc, ConvertD>;

    if (shm_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    kernel<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP, kTM, kTN,
                                         kTK, kTP);
    hipDeviceSynchronize();

    h_d = d_d;

    thrust::host_vector<InType> h_acc(kM * kN * kBatch);
    thrust::fill(h_acc.begin(), h_acc.end(), 0.);
    thrust::device_vector<InType> d_acc = h_acc;

    thrust::host_vector<InType> h_d2(kM * kP * kBatch);
    thrust::fill(h_d2.begin(), h_d2.end(), 0.);
    thrust::device_vector<InType> d_d2 = h_d2;

    cublas_two_gemms(kM, kN, kK, kP, kBatch, A, B, C,
                     thrust::raw_pointer_cast(d_d2.data()),
                     thrust::raw_pointer_cast(d_acc.data()));
    hipDeviceSynchronize();
    h_acc = d_acc;
    h_d2 = d_d2;

    InType* data = thrust::raw_pointer_cast(h_d.data());
    __half* ground_truth = thrust::raw_pointer_cast(h_d2.data());

#ifdef DEBUG
    printf("ours:\n");
    for (int i = 0; i < h_d.size(); ++i) {
        printf("%.3f, ", __half2float(data[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
    printf("\nground_truth:\n");
    for (int i = 0; i < h_d.size(); ++i) {
        printf("%.3f, ", __half2float(ground_truth[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
#endif

    if (check_results(data, ground_truth, kM * kP, epsilon)) {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", passed." << std::endl;
    } else {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", failed." << std::endl;
    }
}

int main() {
    // 在主函数内定义这些变量，而不是全局范围
    using WarpLayout = tl::RowMajor<2, 1>;
    static constexpr int kSharedAccess = 64;

    run<B2BGemmShape<256 /*M*/, 128 /*N*/, 64 /*K*/, 64 /*P*/>,
        B2BGemmShape<64 /*kTM*/, 64 /*kTN*/, 64 /*kTK*/, 64 /*kTP*/>,
        WarpLayout, 1, kSharedAccess>(5e-3);

    return 0;
}
